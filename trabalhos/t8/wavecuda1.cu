#include "hip/hip_runtime.h"


#include <cstdlib>
#include <sys/time.h>
#include <math.h>
#include "wave.h"

//função kernel que faz as frames
__global__
void fazPixel(int width, int frames, unsigned char* pic)
{
  int ix = threadIdx.x;
  int of = blockDim.x;
  for (int frame = ix; frame < frames; frame += ofs) {
  //for (int frame = 0; frame < frames; frame++) {
    for (int row = 0; row < width; row++) {
      for (int col = 0; col < width; col++) {
        float fx = col - 1024/2;
        float fy = row - 1024/2;
        float d = sqrtf( fx * fx + fy * fy );
        unsigned char color = (unsigned char) (160.0f + 127.0f *
                                          cos(d/10.0f - frame/7.0f) /
                                          (d/50.0f + 1.0f));

        pic[frame * width * width + row * width + col] = (unsigned char) color;
      }
    }
  }
}

int main(int argc, char *argv[])
{

  if (argc != 3) {fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]); exit(-1);}
  int width = atoi(argv[1]);
  if (width < 100) {fprintf(stderr, "error: frame_width must be at least 100\n"); exit(-1);}
  int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
  printf("computing %d frames of %d by %d picture\n", frames, width, width);
  
  unsigned char* pic;
  hipMallocManaged(&pic, frames*width*width*sizeof(unsigned char));
    
  // iniciando tempo
  timeval start, end;
  gettimeofday(&start, NULL);

  fazPixel<<<1, frames>>>(width, frames, pic);

  hipDeviceSynchronize();

  // terminando contagem de tempo
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  if ((width <= 256) && (frames <= 100)) {
    for (int frame = 0; frame < frames; frame++) {
      char name[32];
      sprintf(name, "wave%d.bmp", frame + 1000);
      writeBMP(width, width, &pic[frame * width * width], name);
    }
  }

  hipFree(pic);
  return 0;
}